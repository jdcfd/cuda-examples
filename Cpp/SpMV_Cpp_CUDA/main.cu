#include "hip/hip_runtime.h"
/*
Author: Juan D. Colmenares F.
User  : jdcfd@github.com

Sparse Matrix-Vector multiplication in CUDA

Reads in Sparse matrix in MatrixMarket COO format and multiplies
it by a dense vector with random values.

*/
#include <hip/hip_runtime_api.h>
#include <matrix.hpp>
#include <matrix_csr.cuh>
#include <mmio_reader.cuh>
#include <vector_dense.cuh>
#include <hipsparse.h> 

#include <chrono>

#define EPS 1e-08

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

using namespace std;

template <int block_size>
__global__ void sparse_mvm(int * rows, int * cols, double * vals, double * vec, double * res, int nrows, int ncols)
{
    // Block index
    int row = threadIdx.y + blockDim.y*blockIdx.x;
    if(row < nrows){
        int start {rows[row]};
        int end {rows[row+1]}; 
        double sum = 0.0;

        for(int icol = threadIdx.x + start; icol < end; icol += block_size ){
            sum += vals[icol] * vec[cols[icol]];
        }

        // Need to use templated block size to unroll loop
#pragma unroll
        for (int i = block_size >> 1; i > 0; i >>= 1)
            sum += __shfl_down_sync(0xffffffff,sum, i, i*2);

        if(!threadIdx.x){ res[row] = sum; } // write only with first thread        
    }
}


template <int block_size>
__global__ void sparse_mvm_shared(int * rows, int * cols, double * vals, double * vec, double * res, int nrows, int ncols)
{
     // Block index
    int row = threadIdx.y + blockDim.y*blockIdx.x;
    extern __shared__ double sum[];
    if(row < nrows){
        int start {rows[row]};
        int end {rows[row+1]}; 
        sum[block_size*threadIdx.y + threadIdx.x] = 0.0;

        for(int icol = threadIdx.x + start; icol < end; icol += block_size ){
            sum[block_size*threadIdx.y + threadIdx.x] += vals[icol] * vec[cols[icol]];
        }
        // __syncthreads();

        // Need to use templated block size to unroll loop
#pragma unroll
        for (int i = block_size >> 1; i > 0; i >>= 1){
            if(threadIdx.x < i) sum[block_size*threadIdx.y + threadIdx.x] += sum[block_size*threadIdx.y + threadIdx.x + i];
            __syncthreads();
        }

        if(!threadIdx.x){ res[row] = sum[block_size*threadIdx.y + threadIdx.x]; } // write only with first thread        
    }
}


void run_test(int block_size, CSRMatrix *mymat, DenseVector *X, DenseVector *Y, int mnnzpr, bool shared = false){
    // limit the number of threads per row to be no larger than the warp size

    int rows_per_block = 1024 / block_size;
    int num_blocks = (mymat->nrows + rows_per_block - 1) / rows_per_block;
    
    dim3 blocks(num_blocks, 1, 1);
    dim3 threads(block_size, rows_per_block, 1);
    size_t shms = 1024*sizeof(double);

    switch (block_size)
    {
    case 128:
        if(shared){
            sparse_mvm_shared<128><<<blocks,threads,shms>>>(mymat->d_rows, mymat->d_cols, mymat->d_values, 
                                            X->d_val, Y->d_val, mymat->nrows, mymat->ncols);
        }else{
            sparse_mvm<128><<<blocks,threads>>>(mymat->d_rows, mymat->d_cols, mymat->d_values, 
                                            X->d_val, Y->d_val, mymat->nrows, mymat->ncols);
        }
        break;
    case 64:
        if(shared){
            sparse_mvm_shared<64><<<blocks,threads,shms>>>(mymat->d_rows, mymat->d_cols, mymat->d_values, 
                                            X->d_val, Y->d_val, mymat->nrows, mymat->ncols);
        }else{
            sparse_mvm<64><<<blocks,threads>>>(mymat->d_rows, mymat->d_cols, mymat->d_values, 
                                            X->d_val, Y->d_val, mymat->nrows, mymat->ncols);
        }
        break;
    case 32:
        if(shared){
            sparse_mvm_shared<32><<<blocks,threads,shms>>>(mymat->d_rows, mymat->d_cols, mymat->d_values, 
                                            X->d_val, Y->d_val, mymat->nrows, mymat->ncols);
        }else{
            sparse_mvm<32><<<blocks,threads>>>(mymat->d_rows, mymat->d_cols, mymat->d_values, 
                                            X->d_val, Y->d_val, mymat->nrows, mymat->ncols);
        }
        break;
    case 16:
        if(shared){
            sparse_mvm_shared<16><<<blocks,threads,shms>>>(mymat->d_rows, mymat->d_cols, mymat->d_values, 
                                            X->d_val, Y->d_val, mymat->nrows, mymat->ncols);
        }else{
            sparse_mvm<16><<<blocks,threads>>>(mymat->d_rows, mymat->d_cols, mymat->d_values, 
                                            X->d_val, Y->d_val, mymat->nrows, mymat->ncols);
        }
        break;
    case 8:
        if(shared){
            sparse_mvm_shared<8><<<blocks,threads,shms>>>(mymat->d_rows, mymat->d_cols, mymat->d_values, 
                                            X->d_val, Y->d_val, mymat->nrows, mymat->ncols);
        }else{
            sparse_mvm<8><<<blocks,threads>>>(mymat->d_rows, mymat->d_cols, mymat->d_values, 
                                            X->d_val, Y->d_val, mymat->nrows, mymat->ncols);
        }
        break;
    case 4:
        sparse_mvm<4><<<blocks,threads>>>(mymat->d_rows, mymat->d_cols, mymat->d_values, 
                                            X->d_val, Y->d_val, mymat->nrows, mymat->ncols);
        break;
    case 2:
        sparse_mvm<2><<<blocks,threads>>>(mymat->d_rows, mymat->d_cols, mymat->d_values, 
                                            X->d_val, Y->d_val, mymat->nrows, mymat->ncols);
        break;
    default:
        sparse_mvm<1><<<blocks,threads>>>(mymat->d_rows, mymat->d_cols, mymat->d_values, 
                                            X->d_val, Y->d_val, mymat->nrows, mymat->ncols);
        break;
    }
}

int main(int argc, char const *argv[]) {

    if( argc < 2 ){
        cout << "Usage: ./vector_csr <matrix market file>" << endl;
        return -1;
    }

    int ierr {};

    string filename {string(argv[1])};

    // int ntrials {atoi(argv[2])};

    CSRMatrix *mymat {}; 

    CSRMatrixReader reader(filename);

    ierr = reader.mm_init_csr(&mymat); // allocate memory

    if(ierr){
        cout << "Error" << ierr << endl;
        return ierr;
    }

    int mnnzpr = reader.mm_read_csr(mymat); //read from file and convert from coo to csr
    int avgnnzpr = mymat->nnz/mymat->nrows;
    cout << "nrows, ncols, nnz: " << mymat->nrows << ' ' << mymat->ncols << ' '  << mymat->nnz << endl;
    cout << "mnnzpr: " << mnnzpr << endl;
    cout << "avg nnzpr: " << avgnnzpr << endl;

    // mymat->print(); // Print all values. Commented out for large matrices.

    DenseVector X(mymat->ncols);

    X.generate(); // Fill with random numbers 

    DenseVector Y(mymat->ncols); // Initialize with zeros

    // X.print();
    // Y.print();

    // Using functional programming for mat mult to avoid operator overloading
    // No Need for warmup since threads have been used before to intialize vars

    for( int bs = 128; bs > 2; bs >>= 1){
        if(avgnnzpr > bs){
            run_test(bs,mymat,&X,&Y,mnnzpr); 
            run_test(bs,mymat,&X,&Y,mnnzpr,true); 
        }
    }

    Y.update_host(); // only comparing results from last test
    
    // Y.print();

    DenseVector Ycsp(mymat->ncols); // Initialize with zeros

    // Use cuSparse
    // CUSPARSE APIs
    {
        hipsparseHandle_t     handle = NULL;
        hipsparseSpMatDescr_t matA;
        hipsparseDnVecDescr_t vecX, vecY;
        void*                dBuffer    = NULL;
        size_t               bufferSize = 0;
        double alpha = 1.0;
        double beta  = 0.0;
        CHECK_CUSPARSE( hipsparseCreate(&handle) )
        // Create sparse matrix A in CSR format
        CHECK_CUSPARSE( hipsparseCreateCsr(&matA, mymat->nrows, mymat->ncols, mymat->nnz,
                                          mymat->d_rows, mymat->d_cols, mymat->d_values,
                                          HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                          HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
        // Create dense vector X
        CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, mymat->ncols, X.d_val, HIP_R_64F) )
        // Create dense vector y
        CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, mymat->nrows, Ycsp.d_val, HIP_R_64F) )
        // allocate an external buffer if needed
        CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                     handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                     HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize) )
        checkCudaErrors( hipMalloc(&dBuffer, bufferSize) );

        // execute SpMV
        CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                     HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )

        // destroy matrix/vector descriptors
        CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
        CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
        CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
        CHECK_CUSPARSE( hipsparseDestroy(handle) )
    }

    Ycsp.update_host();
    // Ycsp.print();
        
    bool issame {true};    

    for( int i {}; i < Y.size; i++ ){
        issame *= ( fabs(Y.h_val[i] - Ycsp.h_val[i]) < EPS );
    }

    if(issame){
        cout << "Results are correct!" << endl;
    } else {
        cout << "Results are Wrong!" << endl;

        for(int i = 0; i < Y.size ; i++){
            if( fabs(Y.h_val[i] - Ycsp.h_val[i]) >= EPS )
                cout << i << ", Y: " << Y.h_val[i] << ",  Ycsp: " << Ycsp.h_val[i] << endl;
        }
    }

    delete mymat; // Calls destroyer

    mymat = nullptr; 

    return ierr;
}