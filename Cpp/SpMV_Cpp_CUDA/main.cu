#include "hip/hip_runtime.h"
/*
Author: Juan D. Colmenares F.
User  : jdcfd@github.com

Sparse Matrix-Vector multiplication in CUDA

Reads in Sparse matrix in MatrixMarket COO format and multiplies
it by a dense vector with random values.

*/
#include <hip/hip_runtime_api.h>
#include <matrix.hpp>
#include <matrix_csr.cuh>
#include <mmio_reader.cuh>
#include <vector_dense.cuh>
#include <hipsparse.h> 

#define EPS 1e-10

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

using namespace std;

template <int block_size>
__global__ void sparse_mvm(int * rows, int * cols, double * vals, double * vec, double * res, int nrows, int ncols)
{
    // Block index
    int row = threadIdx.y + blockDim.y*blockIdx.x;
    if(row < nrows){
        int start {rows[row]};
        int end {rows[row+1]}; 
        double sum = 0.0;

        for(int icol = threadIdx.x + start; icol < end; icol += block_size ){
            sum += vals[icol] * vec[cols[icol]];
        }

        // Need to use templated block size to unroll loop
#pragma unroll
        for (int i = block_size >> 1; i > 0; i >>= 1)
            sum += __shfl_down_sync(0xffffffff,sum, i, block_size);

        if(!threadIdx.x){ res[row] = sum; } // write only with first thread        
    }
}

/*
template <int block_size>
__global__ void sparse_mvm_shared(int * rows, int * cols, double * vals, double * vec, double * res, int nrows, int ncols)
{
    // Block index
    int row = threadIdx.y + blockDim.y*blockIdx.x;
    double shared [block_size];
    if(row < nrows){
        int start {rows[row]};
        int end {rows[row+1]}; 
        double sum = 0.0;

        for(int icol = threadIdx.x + start; icol < end; icol += block_size ){
            sum += vals[icol] * vec[cols[icol]];
        }

        // Need to use templated block size to unroll loop
#pragma unroll
        for (int i = block_size >> 1; i > 0; i >>= 1)
            sum += __shfl_down_sync(0xffffffff,sum, i, block_size);

        if(!threadIdx.x){ res[row] = sum; } // write only with first thread        
    }
}

template<int block_size>
__global__ void compare_vals(double * val1, double * val2, bool * issame, int nval){
    __shared__ bool redux[block_size];
    int tid = threadIdx.x;
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    redux[tid] = true;
    if( id < nval ){
        redux[tid] = fabs(val1[id] - val2[id]) < 1.0e-10;
        // printf("boolean true: %s\n", redux[tid] ? "true" : "false"); 
    }

    __syncthreads();
    
    for( int s = block_size >> 1; s > 32; s >>= 1 ){
        // threads in warp are synchronized
        if(tid < s) redux[tid] *= redux[tid + s];
        __syncthreads();
    }

#pragma unroll
    for( int s = 32; s > 0; s >>= 1 ){
        // threads in warp are synchronized
        if(tid < s) redux[tid] *= redux[tid + s];
    }

    if(!tid){
        issame[blockIdx.x] = redux[tid];
    }
}
*/

int main(int argc, char const *argv[]) {

    if( argc < 2 ){
        cout << "Usage: ./vector_csr <matrix market file>" << endl;
        return -1;
    }

    int ierr {};

    string filename {string(argv[1])};

    // int ntrials {atoi(argv[2])};

    CSRMatrix *mymat {}; 

    // mymat->print(); //segfaults

    CSRMatrixReader reader(filename);

    ierr = reader.mm_init_csr(&mymat); // allocate memory

    if(ierr){
        cout << "Error" << ierr << endl;
        return ierr;
    }

    int mnnzpr = reader.mm_read_csr(mymat); //read from file and convert from coo to csr

    cout << "mnnzpr: " << mnnzpr << endl;

    // mymat->print(); // Print all values. Commented out for large matrices.

    DenseVector X(mymat->ncols);

    X.generate(); // Fill with random numbers 

    DenseVector Y(mymat->ncols); // Initialize with zeros

    // X.print();
    // Y.print();


    // Using functional programming for mat mult to avoid operator overloading

    // limit the number of threads per row to be no larger than the wavefront (warp) size
    int block_size {32};
    while(block_size > mnnzpr){
        block_size >>= 1;
    }

    int rows_per_block = 1024 / block_size;
    int num_blocks = (mymat->nrows + rows_per_block - 1) / rows_per_block;
    
    dim3 blocks(num_blocks, 1, 1);
    dim3 threads(block_size, rows_per_block, 1);

    switch (block_size)
    {
    case 128:
        sparse_mvm<128><<<blocks,threads>>>(mymat->rows, mymat->cols, mymat->values, 
                                            X.val, Y.val, mymat->nrows, mymat->ncols);
        break;
    case 64:
        sparse_mvm<64><<<blocks,threads>>>(mymat->rows, mymat->cols, mymat->values, 
                                            X.val, Y.val, mymat->nrows, mymat->ncols);
        break;
    case 32:
        sparse_mvm<32><<<blocks,threads>>>(mymat->rows, mymat->cols, mymat->values, 
                                            X.val, Y.val, mymat->nrows, mymat->ncols);
        break;
    case 16:
        sparse_mvm<16><<<blocks,threads>>>(mymat->rows, mymat->cols, mymat->values, 
                                            X.val, Y.val, mymat->nrows, mymat->ncols);
        break;
    case 8:
        sparse_mvm<8><<<blocks,threads>>>(mymat->rows, mymat->cols, mymat->values, 
                                            X.val, Y.val, mymat->nrows, mymat->ncols);
        break;
    case 4:
        sparse_mvm<4><<<blocks,threads>>>(mymat->rows, mymat->cols, mymat->values, 
                                            X.val, Y.val, mymat->nrows, mymat->ncols);
        break;
    case 2:
        sparse_mvm<2><<<blocks,threads>>>(mymat->rows, mymat->cols, mymat->values, 
                                            X.val, Y.val, mymat->nrows, mymat->ncols);
        break;
    default:
        sparse_mvm<1><<<blocks,threads>>>(mymat->rows, mymat->cols, mymat->values, 
                                            X.val, Y.val, mymat->nrows, mymat->ncols);
        break;
    }

    checkCudaErrors( hipDeviceSynchronize() );
    
    // Y.print();

    DenseVector Ycsp(mymat->ncols); // Initialize with zeros

    // Use cuSparse
    // CUSPARSE APIs
    {
        hipsparseHandle_t     handle = NULL;
        hipsparseSpMatDescr_t matA;
        hipsparseDnVecDescr_t vecX, vecY;
        void*                dBuffer    = NULL;
        size_t               bufferSize = 0;
        double alpha = 1.0;
        double beta  = 0.0;
        CHECK_CUSPARSE( hipsparseCreate(&handle) )
        // Create sparse matrix A in CSR format
        CHECK_CUSPARSE( hipsparseCreateCsr(&matA, mymat->nrows, mymat->ncols, mymat->nnz,
                                          mymat->rows, mymat->cols, mymat->values,
                                          HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                          HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
        // Create dense vector X
        CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, mymat->ncols, X.val, HIP_R_64F) )
        // Create dense vector y
        CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, mymat->nrows, Ycsp.val, HIP_R_64F) )
        // allocate an external buffer if needed
        CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                     handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                     HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize) )
        checkCudaErrors( hipMalloc(&dBuffer, bufferSize) );

        // execute SpMV
        CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                     HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )

        // destroy matrix/vector descriptors
        CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
        CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
        CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
        CHECK_CUSPARSE( hipsparseDestroy(handle) )
    }

    // Ycsp.print();
    
    // block_size = 128;
    // int nblocks = (Y.size + block_size - 1)/block_size;
    // bool * issame_pblock;
    // checkCudaErrors( hipMallocManaged((void **)&issame_pblock, nblocks*sizeof(bool)) );

    // // hard code block size for now
    // compare_vals<128><<<nblocks,block_size>>>(Y.val,Ycsp.val,issame_pblock, Y.size);
    
    bool issame {true};    
    checkCudaErrors( hipDeviceSynchronize() );
    
    // for( int i = 0; i < nblocks; ++i){
    //     issame *= issame_pblock[i];
    //     // cout << issame_pblock[i] << endl;
    // }

    for( int i {}; i < Y.size; i++ ){
        issame *= ( fabs(Y.val[i] - Ycsp.val[i]) < EPS );
    }

    if(issame){
        cout << "Results are correct!" << endl;
    } else {
        cout << "Results are Wrong!" << endl;

        for(int i = 0; i < Y.size ; i++){
            if( fabs(Y.val[i] - Ycsp.val[i]) >= EPS )
                cout << i << ", Y: " << Y.val[i] << ",  Ycsp: " << Ycsp.val[i] << endl;
        }
    }

    // checkCudaErrors( hipFree(issame_pblock));

    delete mymat; // Calls destroyer

    mymat = nullptr; 

    return ierr;
}