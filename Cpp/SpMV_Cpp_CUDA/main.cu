#include "hip/hip_runtime.h"
/*
Author: Juan D. Colmenares F.
User  : jdcfd@github.com

Sparse Matrix-Vector multiplication in CUDA

Reads in Sparse matrix in MatrixMarket COO format and multiplies
it by a dense vector with random values.

*/
#include <hip/hip_runtime_api.h>
#include <matrix.hpp>
#include <matrix_csr.cuh>
#include <mmio_reader.cuh>
#include <vector_dense.cuh>
#include <hipsparse.h> 

#include <chrono>

#define TIME_KERNEL(func)                                                   \
{                                                                           \
    auto t0 = std::chrono::high_resolution_clock::now();                    \
    (func);                                         \
    hipDeviceSynchronize();                                                \
    auto t1 = std::chrono::high_resolution_clock::now();                    \
    auto timing = chrono::duration_cast<chrono::nanoseconds>(t1 - t0).count() * 1.e-6; \
    std::cout << "-- Kernel duration: " <<  timing << " ms" << std::endl; \
}

#define EPS 1e-08

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

using namespace std;

template <int block_size>
__global__ void sparse_mvm(int * rows, int * cols, double * vals, double * vec, double * res, int nrows, int ncols)
{
    // Block index
    int row = threadIdx.y + blockDim.y*blockIdx.x;
    if(row < nrows){
        int start {rows[row]};
        int end {rows[row+1]}; 
        double sum = 0.0;

        for(int icol = threadIdx.x + start; icol < end; icol += block_size ){
            sum += vals[icol] * vec[cols[icol]];
        }

        // Need to use templated block size to unroll loop
#pragma unroll
        for (int i = block_size >> 1; i > 0; i >>= 1)
            sum += __shfl_down_sync(0xffffffff,sum, i, i*2);

        if(!threadIdx.x){ res[row] = sum; } // write only with first thread        
    }
}


template <int block_size>
__device__ void warpReduce(volatile double *sdata, unsigned int tid) {
    if (block_size >= 64) sdata[tid] += sdata[tid + 32];
    if (block_size >= 32) sdata[tid] += sdata[tid + 16];
    if (block_size >= 16) sdata[tid] += sdata[tid + 8];
    if (block_size >= 8) sdata[tid] += sdata[tid + 4];
    if (block_size >= 4) sdata[tid] += sdata[tid + 2];
    if (block_size >= 2) sdata[tid] += sdata[tid + 1];
}

template <int block_size>
__global__ void sparse_mvm_shared(int * rows, int * cols, double * vals, double * vec, double * res, int nrows, int ncols)
{
     // Block index
    int row = threadIdx.y + blockDim.y*blockIdx.x;
    extern __shared__ double sum[];
    unsigned int tid = threadIdx.x;

    if(row < nrows){
        int start {rows[row]};
        int end {rows[row+1]}; 
        int icol = tid + start;
        
        if(icol < end){sum[threadIdx.y*(block_size) + tid] = vals[icol] * vec[cols[icol]];}
        else{ sum[threadIdx.y*(block_size) + tid] = 0.0; }

        for( icol = icol + block_size; icol < end; icol+= block_size ){
            sum[threadIdx.y*(block_size) + tid] += vals[icol] * vec[cols[icol]];
        }
        __syncthreads();

        // #pragma unroll
        // for (int i = block_size >> 1; i > 0; i >>= 1){
        //     if(tid < i) sum[threadIdx.y*(block_size) + tid] += sum[threadIdx.y*(block_size) + tid + i];
        //     __syncthreads();
        // }

        if (block_size >= 512) { if (tid < 256) { 
            sum[threadIdx.y*(block_size) + tid] += sum[threadIdx.y*(block_size) + tid + 256]; } 
            __syncthreads(); }
        if (block_size >= 256) { if (tid < 128) { 
            sum[threadIdx.y*(block_size) + tid] += sum[threadIdx.y*(block_size) + tid + 128]; } 
            __syncthreads(); }
        if (block_size >= 128) { if (tid < 64) { 
            sum[threadIdx.y*(block_size) + tid] += sum[threadIdx.y*(block_size) + tid + 64]; } 
            __syncthreads(); }

        if (tid < 32) warpReduce<block_size>(&(sum[threadIdx.y*(block_size)]), tid);

        if(!tid){ res[row] = sum[threadIdx.y*(block_size) + tid]; } // write only with first thread        
    }
}

void compare_values(DenseVector *Y, DenseVector *Yref)
{
    bool issame {true};

    for( int i {}; i < Y->size; i++ ){
        issame *= ( fabs(Y->h_val[i] - Yref->h_val[i]) < EPS*fabs(Yref->h_val[i]) );
    }

    if(issame){
        cout << "-- Results are correct!" << endl;
    } else {
        cout << "-- Results are Wrong!" << endl;

        /*********
        for(int i = 0; i < Y->size ; i++){
            if( fabs(Y->h_val[i] - Yref->h_val[i]) >= EPS )
                cout << i << ", Y: " << Y->h_val[i] << ",  Ycsp: " << Yref->h_val[i] << endl;
        }
        *********/
    }
}

void run_test(int block_size, CSRMatrix *mymat, DenseVector *X, DenseVector *Y, int mnnzpr, bool shared = false){
    // limit the number of threads per row to be no larger than the warp size

    int rows_per_block = 256 / block_size;
    int num_blocks = (mymat->nrows + rows_per_block - 1) / rows_per_block;
    
    dim3 blocks(num_blocks, 1, 1);
    dim3 threads(block_size, rows_per_block, 1);
    size_t shms = rows_per_block*(block_size)*sizeof(double);

    cout << "Running test with block_size=" << block_size << " and shared=" << (shared ? "true" : "false") << endl;

    switch (block_size)
    {
    case 128:
        if(shared){
            TIME_KERNEL((sparse_mvm_shared<128><<<blocks,threads,shms>>>(mymat->d_rows, mymat->d_cols, mymat->d_values,X->d_val, Y->d_val, mymat->nrows, mymat->ncols)))
        }else{
            TIME_KERNEL((sparse_mvm<128><<<blocks,threads>>>(mymat->d_rows, mymat->d_cols, mymat->d_values,X->d_val, Y->d_val, mymat->nrows, mymat->ncols)))
        }
        break;
    case 64:
        if(shared){
            TIME_KERNEL((sparse_mvm_shared<64><<<blocks,threads,shms>>>(mymat->d_rows, mymat->d_cols, mymat->d_values,X->d_val, Y->d_val, mymat->nrows, mymat->ncols)))
        }else{
            TIME_KERNEL((sparse_mvm<64><<<blocks,threads>>>(mymat->d_rows, mymat->d_cols, mymat->d_values,X->d_val, Y->d_val, mymat->nrows, mymat->ncols)))
        }
        break;
    case 32:
        if(shared){
            TIME_KERNEL((sparse_mvm_shared<32><<<blocks,threads,shms>>>(mymat->d_rows, mymat->d_cols, mymat->d_values,X->d_val, Y->d_val, mymat->nrows, mymat->ncols)))
        }else{
            TIME_KERNEL((sparse_mvm<32><<<blocks,threads>>>(mymat->d_rows, mymat->d_cols, mymat->d_values,X->d_val, Y->d_val, mymat->nrows, mymat->ncols)))
        }
        break;
    case 16:
        if(shared){
            TIME_KERNEL((sparse_mvm_shared<16><<<blocks,threads,shms>>>(mymat->d_rows, mymat->d_cols, mymat->d_values,X->d_val, Y->d_val, mymat->nrows, mymat->ncols)))
        }else{
            TIME_KERNEL((sparse_mvm<16><<<blocks,threads>>>(mymat->d_rows, mymat->d_cols, mymat->d_values,X->d_val, Y->d_val, mymat->nrows, mymat->ncols)))
        }
        break;
    case 8:
        if(shared){
            TIME_KERNEL((sparse_mvm_shared<8><<<blocks,threads,shms>>>(mymat->d_rows, mymat->d_cols, mymat->d_values,X->d_val, Y->d_val, mymat->nrows, mymat->ncols)))
        }else{
            TIME_KERNEL((sparse_mvm<8><<<blocks,threads>>>(mymat->d_rows, mymat->d_cols, mymat->d_values,X->d_val, Y->d_val, mymat->nrows, mymat->ncols)))
        }
        break;
    case 4:
        if(shared){
            TIME_KERNEL((sparse_mvm_shared<4><<<blocks,threads,shms>>>(mymat->d_rows, mymat->d_cols, mymat->d_values,X->d_val, Y->d_val, mymat->nrows, mymat->ncols)))
        }else{
            TIME_KERNEL((sparse_mvm<4><<<blocks,threads>>>(mymat->d_rows, mymat->d_cols, mymat->d_values,X->d_val, Y->d_val, mymat->nrows, mymat->ncols)))
        }
        break;
    case 2:
        TIME_KERNEL((sparse_mvm<2><<<blocks,threads>>>(mymat->d_rows, mymat->d_cols, mymat->d_values,X->d_val, Y->d_val, mymat->nrows, mymat->ncols)))
        break;
    default:
        TIME_KERNEL((sparse_mvm<1><<<blocks,threads>>>(mymat->d_rows, mymat->d_cols, mymat->d_values,X->d_val, Y->d_val, mymat->nrows, mymat->ncols)))
        break;
    }
}

int main(int argc, char const *argv[]) {

    if( argc < 2 ){
        cout << "Usage: ./vector_csr <matrix market file>" << endl;
        return -1;
    }

    int ierr {};

    string filename {string(argv[1])};

    // int ntrials {atoi(argv[2])};

    CSRMatrix *mymat {}; 

    CSRMatrixReader reader(filename);

    ierr = reader.mm_init_csr(&mymat); // allocate memory

    if(ierr){
        cout << "Error" << ierr << endl;
        return ierr;
    }

    int mnnzpr = reader.mm_read_csr(mymat); //read from file and convert from coo to csr
    int avgnnzpr = mymat->nnz/mymat->nrows;
    cout << "nrows, ncols, nnz: " << mymat->nrows << ' ' << mymat->ncols << ' '  << mymat->nnz << endl;
    cout << "mnnzpr: " << mnnzpr << endl;
    cout << "avg nnzpr: " << avgnnzpr << endl;

    // mymat->print(); // Print all values. Commented out for large matrices.

    DenseVector X(mymat->ncols);

    X.generate(); // Fill with random numbers 

    DenseVector Y(mymat->ncols); // Initialize with zeros

    // X.print();
    // Y.print();

    DenseVector Ycsp(mymat->ncols); // Initialize with zeros

    // Use cuSparse
    // CUSPARSE APIs
    {
        hipsparseHandle_t     handle = NULL;
        hipsparseSpMatDescr_t matA;
        hipsparseDnVecDescr_t vecX, vecY;
        void*                dBuffer    = NULL;
        size_t               bufferSize = 0;
        double alpha = 1.0;
        double beta  = 0.0;

        CHECK_CUSPARSE( hipsparseCreate(&handle) )
        // Create sparse matrix A in CSR format
        CHECK_CUSPARSE( hipsparseCreateCsr(&matA, mymat->nrows, mymat->ncols, mymat->nnz,
                                          mymat->d_rows, mymat->d_cols, mymat->d_values,
                                          HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                          HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
        // Create dense vector X
        CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, mymat->ncols, X.d_val, HIP_R_64F) )
        // Create dense vector y
        CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, mymat->nrows, Ycsp.d_val, HIP_R_64F) )
        
        // allocate an external buffer if needed      
        CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                     handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                     HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize) )
        
        cout << "Buffer size: " << bufferSize << endl; 
        checkCudaErrors( hipMalloc(&dBuffer, bufferSize) );

        // execute SpMV      
        auto t0 = std::chrono::high_resolution_clock::now();    
        CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                     HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )
        auto t1 = std::chrono::high_resolution_clock::now();
        
        auto timing = chrono::duration_cast<chrono::nanoseconds>(t1 - t0).count() * 1.e-6; \
        cout << endl << "-- hipsparseSpMV duration: " <<  timing << " ms" << endl << endl;

        // destroy matrix/vector descriptors
        CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
        CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
        CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
        CHECK_CUSPARSE( hipsparseDestroy(handle) )
    }

    Ycsp.update_host();

    checkCudaErrors(hipDeviceSetSharedMemConfig ( hipSharedMemBankSizeEightByte ));

    for( int bs = 4; bs < avgnnzpr; bs *= 2){
        if(true){
            // reset Y
            Y.fill(0.0);
            run_test(bs,mymat,&X,&Y,mnnzpr,false); 
            Y.update_host(); // only comparing results from last test
            compare_values(&Y, &Ycsp);

            Y.fill(0.0);
            run_test(bs,mymat,&X,&Y,mnnzpr,true); 
            Y.update_host(); // only comparing results from last test
            compare_values(&Y, &Ycsp);
            cout << endl;
        }
    }

    delete mymat; // Calls destroyer

    mymat = nullptr; 

    return ierr;
}