#include <matrix.hpp>
#include <matrix_csr.cuh>
#include <iostream>
#include <hip/hip_runtime_api.h>

CSRMatrix::CSRMatrix(int nr, int nc, int nnz) : MatrixBase(nr,nc){
    if( nnz > nr*nc ){
        throw std::invalid_argument( "received nnz > nrows * ncols" );
    }
    this->nnz = nnz;
    this->rows = new int[nr+1]{};
    this->cols = new int[nnz]{};
    this->values = new double[nnz]{};

    checkCudaErrors(hipMalloc(reinterpret_cast<void **> (&(this->d_rows)),sizeof(int)*(nr+1)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **> (&(this->d_cols)),sizeof(int)*(nnz+1)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **> (&(this->d_values)),sizeof(double)*(nnz+1)));
}

CSRMatrix::~CSRMatrix(){
    delete [] this->rows; this->rows = nullptr;
    delete [] this->cols; this->cols = nullptr;
    delete [] this->values; this->values = nullptr;
    checkCudaErrors(hipFree(this->d_rows));
    checkCudaErrors(hipFree(this->d_cols));
    checkCudaErrors(hipFree(this->d_values));
    this->d_rows = nullptr;
    this->d_cols = nullptr;
    this->d_values = nullptr;
}

void CSRMatrix::update_device(){
    size_t size_rows {sizeof(int)*(this->nrows+1)};
    size_t size_cols {sizeof(int)*(this->nnz)};
    size_t size_vals {sizeof(double)*(this->nnz)};

    checkCudaErrors(hipMemcpy(this->d_rows  , this->rows  , size_rows, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(this->d_cols  , this->cols  , size_cols, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(this->d_values, this->values, size_vals, hipMemcpyHostToDevice));
}

void CSRMatrix::update_host(){
    size_t size_rows {sizeof(int)*(this->nrows+1)};
    size_t size_cols {sizeof(int)*(this->nnz)};
    size_t size_vals {sizeof(double)*(this->nnz)};

    checkCudaErrors(hipMemcpy(this->rows  , this->d_rows  , size_rows, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(this->cols  , this->d_cols  , size_cols, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(this->values, this->d_values, size_vals, hipMemcpyDeviceToHost));
}

void CSRMatrix::print(){
    this->update_host();
    if(this->nrows > 0 && this->nnz > 0){
        std::cout << "Nrows: " << this->nrows << " Ncols: " << this->ncols << std::endl;
        std::cout << "Nnz: "   << this->nnz << std::endl;
        for(int i {}; i < this->nrows + 1; i++){
            std::cout << "rows[" << i << "] = " << this->rows[i] << std::endl;
        }
        for(int i {}; i < this->nnz; i++){
            std::cout << "cols[" << i << "]= " << this->cols[i] << ", val[" << i << "]= " << this->values[i] << std::endl;
        }
        std::cout << std::endl;
    }else{
        std::cout << "Matrix has not been set." << std::endl;
    }
}